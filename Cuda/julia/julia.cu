#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>

#include <iostream>
#include <cstring>

using namespace std;

using ui32 = unsigned int;

ui32 const dim = 2048;

struct complex {
  float r; float i;
  __device__ complex(float r, float i) : r(r), i(i) {}
  __device__ float magnitude() {return r*r + i*i;}
  __device__ complex operator*(const complex& c) {
    return complex(r * c.r - i * c.i, i * c.r + r * c.i);
  }
  __device__ complex operator+(const complex& c) {
    return complex(r + c.r, i + c.i);
  }
};

__device__ unsigned char julia( int x, int y )
{
  const float scale = 1.5;

  float jx = scale * (float)(dim/2.0f - x)/(dim/2.0f);
  float jy = scale * (float)(dim/2.0f - y)/(dim/2.0f);

  ::complex c(-0.8, 0.156);
  ::complex a(jx, jy);

  for(unsigned int i = 0 ; i < 200 ; i+=2) {
    a = a * a + c;
    if(a.magnitude() > 1000) {
      return 0;
    }
  }
  return 255;
}

__global__ void fractal( unsigned char * out )
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  
  out[ i + j * dim ] = julia( i, j );
}


int main()
{
  std::size_t const size = dim * dim;
  unsigned char * out_h;
  unsigned char * out_d;
  
  hipHostMalloc( &out_h, size );
  hipMalloc( &out_d, size );

  dim3 block( 32, 32 );
  dim3 grid( (dim-1)/block.x + 1, (dim-1)/block.y + 1 );

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start );

  fractal<<< grid, block >>>( out_d );
  hipDeviceSynchronize();
  auto err = hipGetLastError();
  if( err != hipSuccess )
  {
    std::cerr << hipGetErrorString( err ) << std::endl;
  }
  
  
  hipMemcpy( out_h, out_d, size, hipMemcpyDeviceToHost );

  hipDeviceSynchronize();
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";
  
  cv::Mat m_out( dim, dim, CV_8UC1, out_h );
  
  imwrite( "julia.png", m_out );

  return 0;
}

