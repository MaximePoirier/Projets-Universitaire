#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <math.h>
#include <sys/time.h>

__global__ void grayscaleSobel( unsigned char * rgb, unsigned char * g,unsigned char * out, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * (blockDim.x-2) +threadIdx.x;
  auto j = blockIdx.y * (blockDim.y-2) +threadIdx.y;
  int h, v, res;
  extern __shared__ int s[];

  if( i < cols && j < rows ) {
    s[ threadIdx.y * blockDim.x + threadIdx.x ] = (
      307 * rgb[ 3 * ( j * cols + i ) + 0]
      + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
      + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
    ) / 1024;
  }

  __syncthreads();

  if( threadIdx.y < blockDim.x-1 && threadIdx.x < blockDim.y-1  && threadIdx.x > 0 && threadIdx.y > 0) {
		// Horizontal
		h =     s[((threadIdx.y - 1) * blockDim.x + threadIdx.x - 1)] -     s[((threadIdx.y - 1) * blockDim.x + threadIdx.x + 1)]
		  + 2 * s[( threadIdx.y      * blockDim.x + threadIdx.x - 1)] - 2 * s[( threadIdx.y      * blockDim.x + threadIdx.x + 1)]
		  +     s[((threadIdx.y + 1) * blockDim.x + threadIdx.x - 1)] -     s[((threadIdx.y + 1) * blockDim.x + threadIdx.x + 1)];

		// Vertical

		v =     s[((threadIdx.y - 1) * blockDim.x + threadIdx.x - 1)] -     s[((threadIdx.y + 1) * blockDim.x + threadIdx.x - 1)]
		  + 2 * s[((threadIdx.y - 1) * blockDim.x + threadIdx.x    )] - 2 * s[((threadIdx.y + 1) * blockDim.x + threadIdx.x    )]
		  +     s[((threadIdx.y - 1) * blockDim.x + threadIdx.x + 1)] -     s[((threadIdx.y + 1) * blockDim.x + threadIdx.x + 1)];

		//h = h > 255 ? 255 : h;
		//v = v > 255 ? 255 : v;
		res = h*h + v*v;
		res = res > 255*255 ? res = 255*255 : res;

		out[(j * cols + i)] = sqrt(float(res));
	}
}


 void init(std::string const & name_file_in ,std::string const & name_file_out,int nthreadsx,int nthreadsy){
        cv::Mat m_in = cv::imread(name_file_in, cv::IMREAD_UNCHANGED );

   auto rgb = m_in.data;
   auto rows = m_in.rows;
   auto cols = m_in.cols;

   std::vector< unsigned char > g( rows * cols ); // image de sortie.
   std::vector< unsigned char > out( rows * cols );//image sortie sobel

   cv::Mat m_out( rows, cols, CV_8UC1, out.data() );

   unsigned char * rgb_d;
   unsigned char * g_d;
   unsigned char * out_d;

   hipEvent_t start,stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start);



   hipMalloc( &rgb_d,3*rows * cols *sizeof(unsigned char) ); // allocation pour l'image d'entrée sur le device.
   hipMalloc( &g_d,g.size()*sizeof(unsigned char) ); // allocation pour l'image de sortie gray sur le device.
   hipMalloc( &out_d,out.size()*sizeof(unsigned char) ); // allocation pour l'image de sortie sobel sur le device.

   hipMemcpy(rgb_d,rgb,3*rows * cols *sizeof(unsigned char) , hipMemcpyHostToDevice ); // copie de l'image d'entrée vers le device.

   dim3 t( nthreadsx, nthreadsy );
   dim3 b( ( cols - 1) / (t.x-2) + 1 , ( rows - 1 ) / (t.y-2) + 1 );

   grayscaleSobel
<<< b, t, nthreadsx*nthreadsy *sizeof(int) >>>( rgb_d ,g_d , out_d ,cols,rows );

   hipMemcpy(out.data(),out_d,out.size()*sizeof(unsigned char),hipMemcpyDeviceToHost ); // récupération de l'image
   hipDeviceSynchronize(); // Attente de la fin d'exécution du kernel.
   hipError_t err = hipGetLastError();
   if( err != hipSuccess )
   {
     std::cerr << hipGetErrorString( err ); // récupération du message associé au code erreur.
   }

   hipEventRecord(stop);
   hipEventSynchronize(stop);
   float elapseTime;
   hipEventElapsedTime(&elapseTime,start,stop);
   std::cout<<elapseTime<<"ms"<<std::endl;
   cv::imwrite( name_file_out, m_out ); // sauvegarde de l'image.

   hipFree( rgb_d );
   hipFree( g_d);
   hipFree(out_d);
 }

int main()
{

  init("../in.jpg","../out.jpg",32,32);

 }
