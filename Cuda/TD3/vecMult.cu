#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


__global__ void vecAdd(int * v0, int * v1, int * v2, std::size_t size){
  extern __shared__ int v0tmp[];
  auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid<size){
    v0tmp[tid] = v0[tid];
    v2[tid] = 0;
    for(std::size_t i = 0; i<10; i++){
        v2[tid] += v0tmp[tid] + v1[tid];
    }
  }
}


int main(){
  std::size_t size = 1000;

  std::vector<int> v0(size);
  std::vector<int> v1(size);
  std::vector<int> v2(size);

  int * v0_d = nullptr;
  int * v1_d = nullptr;
  int * v2_d = nullptr;

  for(std::size_t i = 0; i < v0.size(); i++){
    v0[i] = v1[i] = i;
  }

  hipError_t err;

  err = hipMalloc(&v0_d, v0.size() * sizeof(int));
  if(err != hipSuccess){
    std::cerr << hipGetErrorString(err) << std::endl;
    return 1;
  }
  err = hipMalloc(&v1_d, v1.size() * sizeof(int));

  err = hipMalloc(&v2_d,v2.size() * sizeof(int));

  hipMemcpy(v0_d,v0.data(),v0.size() * sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(v1_d,v0.data(),v1.size() * sizeof(int),hipMemcpyHostToDevice);

  dim3 block(1024);
  dim3 grid((size-1)/ block.x +1);
  vecAdd<<<grid,block, size * sizeof(int)/*octets*/>>>(v0_d,v1_d,v2_d,v0.size());

  hipDeviceSynchronize();
  err = hipGetLastError();
  if(err != hipSuccess){
    std::cerr << hipGetErrorString(err) << std::endl;
    return 1;
  }

  hipMemcpy(v2.data(),v2_d,v2.size() * sizeof(int),hipMemcpyDeviceToHost);

  for(std::size_t i = 0; i < v2.size(); i++){
    printf("%d\n",v2[i] );
  }

  hipFree(v0_d);
  hipFree(v1_d);
  hipFree(v2_d);
}
