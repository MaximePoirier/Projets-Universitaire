
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


__global__ void vecadd( int * v0, int * v1, std::size_t size )
{
  auto tid = threadIdx.x;
  v0[ tid ] += v1[ tid ];
}


int main()
{
  hipError_t err;

  std::size_t const size = 100;
  std::size_t const sizeb = size * sizeof( int );

  std::vector< int > v0( size );
  std::vector< int > v1( size );

  /*
  int * v0_h = nullptr;
  int * v1_h = nullptr;
  */

  for( std::size_t i = 0 ; i < size ; ++i )
  {
    v0[ i ] = v1[ i ] = i;
  }

  int * v0_d = nullptr;
  int * v1_d = nullptr;

  hipHostRegister( v0.data(), sizeb, hipHostRegisterDefault );
  hipHostRegister( v1.data(), sizeb, hipHostRegisterDefault );

  /*
  err = cudaMallocHost( &v0_h, sizeb );
  if( err != cudaSuccess ) { std::cerr << "Error" << std::endl; }
  err = cudaMallocHost( &v1_h, sizeb);
  if( err != cudaSuccess ) { std::cerr << "Error" << std::endl; }
  */
  /*
  for( std::size_t i = 0 ; i < size ; ++i )
  {
    v0_h[ i ] = 5;
    v1_h[ i ] = 5;
  }
  */

  err = hipHostMalloc( &v0_d, sizeb , hipHostMallocDefault);
  if( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
  err = hipHostMalloc( &v1_d, sizeb , hipHostMallocDefault);
  if( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

  hipStream_t streams[ 4 ];

  for( std::size_t i = 0 ; i < 4 ; ++i )
  {
    hipStreamCreate( &streams[ i ] );
  }
  for( std::size_t i = 0 ; i < 4 ; ++i )
  {
    err = hipMemcpyAsync( v0_d + i*size/4, v0.data() + i*size/4, sizeb/4, hipMemcpyHostToDevice, streams[ i ] );
    if( err != hipSuccess ) { std::cerr << "Error 3" << std::endl; }

    err = hipMemcpyAsync( v1_d + i*size/4, v1.data() + i*size/4, sizeb/4, hipMemcpyHostToDevice, streams[ i ] );
    if( err != hipSuccess ) { std::cerr << "Error 3.2" << std::endl; }
  }

  for( std::size_t i = 0 ; i < 4 ; ++i )
  {
    vecadd<<< 1, 25, 0, streams[ i ] >>>( v0_d + i*size/4, v1_d + i*size/4, size/4 );
    err = hipGetLastError();
    if( err != hipSuccess ) { std::cerr << "Error 3.5" << std::endl; }
  }

  for( std::size_t i = 0 ; i < 4 ; ++i )
  {
    err = hipMemcpyAsync( v0.data() + i*size/4, v0_d + i*size/4, sizeb/4, hipMemcpyDeviceToHost, streams[ i ] );
    if( err != hipSuccess ) { std::cerr << "Error 4" << std::endl; }
  }
  hipDeviceSynchronize( );

  for( std::size_t i = 0 ; i < 4 ; ++i )
  {
    hipStreamDestroy( streams[ i ] );
  }

  for( auto x: v0 )
  {
    std::cout << x << std::endl;
  }

  return 0;
}
