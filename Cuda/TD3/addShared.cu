#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


__global__ void addShared(int * v0, std::size_t size){
  extern __shared__ int v0tmp[];
  auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  v0tmp[tid] = v0[tid];
  __syncthreads();
  if(tid>0 && tid<size - 1){
    v0tmp[tid] += v0tmp[tid-1] + v0tmp[tid+1];
    v0[tid] = v0tmp[tid];
  }
}


int main(int argc, char const *argv[]) {
  std::size_t size = 1024;

  std::vector<int> v0(size);
  int * v0_d = nullptr;

  for(std::size_t i = 0; i < v0.size(); i++){
    v0[i] = 1;
  }

  hipMalloc(&v0_d, v0.size() * sizeof(int));

  hipMemcpy(v0_d,v0.data(),v0.size() * sizeof(int),hipMemcpyHostToDevice);

  addShared<<<1,1024, 1024 * sizeof(int) >>>(v0_d,v0.size());


  hipDeviceSynchronize();

  hipMemcpy(v0.data(),v0_d,v0.size() * sizeof(int),hipMemcpyDeviceToHost);

  for(std::size_t i = 0; i < v0.size(); i++){
    printf("%d\n",v0[i] );
  }

  hipFree(v0_d);

  return 1;
}
