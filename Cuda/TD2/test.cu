
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

__global__ void fill(float * a0,std::size_t size){
  auto tid = threadIdx.x;
  if(tid < size){
      a0[tid] = 1.0f;
  }
}


int main(){

  float *a0_d = nullptr;

  std::size_t const size =10000000;
  std::vector<float> a0_h(10);

  hipMalloc(&a0_d,size*sizeof(float));

  dim3 block(32);
  dim3 grid((size-1)/block.x + 1);

  fill<<<1,size>>>(a0_d,size);

  hipMemcpy(a0_h.data(),a0_d,size*sizeof(float), hipMemcpyDeviceToHost);

  hipFree(a0_d);

  for(auto v: a0_h){
    std::cout << v << std::endl;
  }

  return 0;
}
