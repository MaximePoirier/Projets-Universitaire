#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

__global__ void fill(int * v,std::size_t size){
  // Get the id of the thread (0 -> 99)
  auto tid = threadIdx.x;
  v[tid] = tid;
}

int main(){
  std::vector<int> v(100);
  int * v_d = nullptr;

  //Allocate on the Device
  hipMalloc(&v_d,v.size()*sizeof(int));

  fill<<<1,100>>>(v_d, v.size());

  hipMemcpy(v.data(),v_d,v.size() * sizeof(int), hipMemcpyDeviceToHost);

  for(auto x: v){
    std::cout<< x <<" ";
  }
}
