#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


__global__ void matAdd(int * m0_d, int * m1_d, std::size_t w, std::size_t h){
  auto tidx = blockIdx.x * blockDim.x + threadIdx.x;
  auto tidy = blockIdx.y * blockDim.y + threadIdx.y;
  if(tidx<w && tidy<h){
    m0_d[tidy * w + tidx] += m1_d[tidy * w + tidx];
  }
}


int main(){
  std::size_t w = 10;
  std::size_t h = 10;
  std::size_t size = w*h;

  std::vector<int> m0_h(size);
  std::vector<int> m1_h(size);

  int * m0_d = nullptr;
  int * m1_d = nullptr;

  for(std::size_t i = 0; i < size; i++){
    m0_h[i] = m1_h[i] = i;
  }

  hipError_t err;

  err = hipMalloc(&m0_d, m0_h.size() * sizeof(int));
  if(err != hipSuccess){
    std::cerr << hipGetErrorString(err) << std::endl;
    return 1;
  }
  err = hipMalloc(&m1_d, m1_h.size() * sizeof(int));

  hipMemcpy(m0_d,m0_h.data(),m0_h.size() * sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(m1_d,m1_h.data(),m1_h.size() * sizeof(int),hipMemcpyHostToDevice);

  dim3 block(32,32);
  dim3 grid((w-1)/ block.x +1, (h-1)/ block.y +1);
  matAdd<<<grid,block>>>(m0_d,m1_d,w,h);

  hipDeviceSynchronize();
  err = hipGetLastError();
  if(err != hipSuccess){
    std::cerr << hipGetErrorString(err) << std::endl;
    return 1;
  }

  hipMemcpy(m0_h.data(),m0_d,m0_h.size() * sizeof(int),hipMemcpyDeviceToHost);

  for(std::size_t i = 0; i < m0_h.size(); i++){
    printf("%d\n",m0_h[i] );
  }

  hipFree(m0_d);
  hipFree(m1_d);
}
