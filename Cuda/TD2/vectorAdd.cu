#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


__global__ void vecAdd(int * v0, int * v1, std::size_t size){
  auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid<size){
    v0[tid] += v1[tid];
  }
}


int main(){
  std::size_t size = 10000;

  std::vector<int> v0(size);
  std::vector<int> v1(size);

  int * v0_d = nullptr;
  int * v1_d = nullptr;

  for(std::size_t i = 0; i < v0.size(); i++){
    v0[i] = v1[i] = i;
  }

  hipError_t err;

  err = hipMalloc(&v0_d, v0.size() * sizeof(int));
  if(err != hipSuccess){
    std::cerr << hipGetErrorString(err) << std::endl;
    return 1;
  }
  err = hipMalloc(&v1_d, v1.size() * sizeof(int));

  hipMemcpy(v0_d,v0.data(),v0.size() * sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(v1_d,v0.data(),v1.size() * sizeof(int),hipMemcpyHostToDevice);

  dim3 block(1024);
  dim3 grid((size-1)/ block.x +1);
  vecAdd<<<grid,block>>>(v0_d,v1_d,v0.size());

  hipDeviceSynchronize();
  err = hipGetLastError();
  if(err != hipSuccess){
    std::cerr << hipGetErrorString(err) << std::endl;
    return 1;
  }

  hipMemcpy(v0.data(),v0_d,v0.size() * sizeof(int),hipMemcpyDeviceToHost);

  for(std::size_t i = 0; i < v0.size(); i++){
    printf("%d\n",v0[i] );
  }

  hipFree(v0_d);
  hipFree(v1_d);
}
