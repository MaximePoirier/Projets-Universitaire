#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <iostream>

__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x*blockDim.x+ threadIdx.x;
  auto j = blockIdx.y*blockDim.y+ threadIdx.y;
  if( i < cols && j < rows ) {
    g[ j * cols + i ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
		       + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
		       + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
		       ) / 1024;
  }
}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );

  auto rgb = m_in.data;

  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( rows * cols ); // image de sortie.

  cv::Mat m_out( rows, cols, CV_8UC1, g.data() );

  unsigned char * rgb_d;
  unsigned char * g_d;
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  hipMalloc(&rgb_d,3*rows*cols); // allocation pour l'image d'entrée sur le device.
  hipMalloc(&g_d,rows*cols); // allocation pour l'image de sortie sur le device.

  hipMemcpy(rgb_d,rgb,3*rows*cols,hipMemcpyHostToDevice); // copie de l'image d'entrée vers le device.

  dim3 t( 32, 32 );
  dim3 b( ( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );
  grayscale<<< b, t >>>( rgb_d,g_d,cols,rows );

  hipMemcpy(g.data(),g_d,rows*cols, hipMemcpyDeviceToHost); // récupération de l'image en niveaux de gris sur l'hôte.

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float elapseTime;
  hipEventElapsedTime(&elapseTime,start,stop);
  std::cout<<elapseTime<<"ms"<<std::endl;
  
  cv::imwrite( "out.jpg", m_out ); // sauvegarde de l'image.
  
  hipFree( rgb_d );
  hipFree( g_d);

  return 0;
}
